
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <cstdio>
#include<stdlib.h>
#include <iostream>
#include <string.h>
using namespace std;
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

const int N = 265;
const int prethread = 256;


__device__ int Max(int a,int b)
{
	return a < b ? b : a;
}

__global__ void cal(int *a,int *t)
{
	__shared__ int share[prethread];
	int pos = threadIdx.x + blockDim.x * blockIdx.x;
	int id = threadIdx.x;
	int temp = 0;
	while(pos < N)
	{
		temp = Max(temp,a[pos]);
		pos += blockDim.x * gridDim.x;
	}
	share[id] = temp;
	__syncthreads();
	int size = blockDim.x /2;

	while(size != 0)
	{
		if(id < size)
			share[id] = Max(share[id],share[id + size]);
		size >>= 1;
	}
	if(id == 0)
		t[blockIdx.x] = share[id];
}

void solve(int *dev_a,int *dev_t)
{
	cal <<<min(32, (N + prethread -1)/ prethread),prethread >>> (dev_a,dev_t);
	int *t =(int *) malloc(sizeof(int) * N);
	hipMemcpy(t,dev_t,sizeof(int )*N ,hipMemcpyDeviceToHost);
	for(int i = 0;i <= min(32, (N + prethread -1));i++)
		cout << t[i] <<" " ;
	cout <<endl;
}

int main()
{
	int *dev_a,*dev_t,Max = 0;
	int *a = (int*)malloc(sizeof(int) * N );
	srand(NULL);
	for(int i = 0;i < N;i++)
	{
		a[i] = rand() % N;
		Max = max(Max,a[i]);
	}
	cout << Max <<endl;
	hipMalloc( (void**)&dev_a,sizeof(int)*N );
	hipMalloc( (void**)&dev_t,sizeof(int)*N );
	hipMemcpy(dev_a,a,sizeof(int )*N ,hipMemcpyHostToDevice);
	cout << "can" <<endl;
	solve(dev_a,dev_t);
    return 0;
}

