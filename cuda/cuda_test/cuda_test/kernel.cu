/*
���ι�Լ������ֵ
*/
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <cstdio>
#include<stdlib.h>
#include <iostream>
#include <string.h>

#define judge(res) if(res!=hipSuccess){exit(-1);}  

using namespace std;
const int maxn = 1000;
const int N = 256;
const int Size = 1000;
__global__ void get_Max(int *ans,int *c)
{
	__shared__ int share[N];
	int id = threadIdx.x;
	int temp = -1,i = id;
	int pos = id + blockDim.x*blockIdx.x;
	int t = pos ;
	if(pos >= Size)
		return ;
	while(pos < Size)
	{
		if(temp < c[pos])
			temp = c[pos];
		pos += gridDim.x*blockDim.x;
	}
	share[id] = temp;
    __syncthreads();

	int len = blockDim.x/2;
	while(len)
	{
		if(id < len && t + len < Size)
		{
			if(share[id] < share[id + len])
			share[id] = share[id + len];
		}
		len >>= 1;
		__syncthreads();
	}
	if(id == 0)
	{
		ans[blockIdx.x] = share[id];
	}
}

__global__ void get_ans(int *ans,int *c,int l)
{
	__shared__ int ta[N];
	int id = threadIdx.x;
	ta[id] = c[id];
	int len = l / 2;
	while(len)
	{
		if(id < len && id+len < l)
		{
			if(ta[id] < ta[id+len])
				ta[id] = ta[id+len]; 
		}
		len >>= 1;
		__syncthreads();
	}
	if(id == 0)
		ans[0] = ta[id];
}

void cal(int *ans,int *toans,int *dev_a)
{
	get_Max<<<N,N>>>(toans,dev_a);
	int *out =(int*)malloc(sizeof(int )*N);
	hipMemcpy(out,toans,sizeof(int)*N,hipMemcpyDeviceToHost);
	printf("The first time:\n");
	for(int i = 0;i < (Size + N -1) / N;i++)
		printf("%d ",out[i]);
	printf("\n");

	cout << "the second time" <<endl;
	int *tans = (int*)malloc(sizeof(int));
	get_ans<<<1,(Size + N -1) / N>>>(ans,toans,(Size + N -1) / N);
	hipMemcpy(tans,ans,sizeof(int),hipMemcpyDeviceToHost);
	printf("%d\n",tans[0]);
}

int main()
{
	int *a,*dev_a;
	int *ans,*tans,*toans;
	freopen("read.txt","r",stdin);

	hipMalloc((void**)&(dev_a),sizeof(int)*Size);
	hipMalloc((void**)&(toans),sizeof(int)*N);
	hipMalloc((void**)&(ans),sizeof(int));
	a = (int*)malloc(sizeof(int) * Size);
	tans = (int*)malloc(sizeof(int));

	hipMemset(toans,0,sizeof(toans));
	hipMemset(ans,0,sizeof(ans));
	memset(a,0,sizeof(a));
	for(int i = 0;i < Size;i++)
		scanf("%d ",&a[i]);
	//for(int i = 0;i < Size;i++)
	//	printf("%d ",a[i]);
	//printf("\n");

    hipMemcpy(dev_a,a,sizeof(int)*Size,hipMemcpyHostToDevice);
	cal(ans,toans,dev_a);
	return 0;
}