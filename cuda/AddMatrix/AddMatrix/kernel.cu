
#include "hip/hip_runtime.h"

#include <hip/device_functions.h>

#include <stdio.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <time.h>
using namespace std ;
const int N = 100;
const int M = 10;

__global__ void add(int *a,  int *b,  int *c)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	c[offset] = a[offset] + b[offset];
}

void random(int *a,int n,int m)
{
	srand(NULL);
	for(int i = 0;i < n*m;i++)
		a[i] = rand()%n;
}

void print(int *a,char op)
{
	printf("Mat %c\n",op);
	for(int i = 0;i < N;i++)
	{
		for(int j = 0;j < M;j++)
			printf("%d ",a[i * M + j]);
	    cout <<endl;
	}
}

int main()
{
	freopen("out.txt","w",stdout);
	int *b = (int *)malloc(sizeof(int) * N * M);
	int *a = (int *)malloc(sizeof(int) * N * M);
	int *c = (int *)malloc(sizeof(int) * N * M);
	random(a,N,M);
	random(b,N,M);
	print(a,'a');
	print(b,'b');
	int *dev_a,*dev_b,*dev_c;
	hipMalloc((void**)&dev_a,sizeof(int) * N * M);
	hipMalloc((void**)&dev_b,sizeof(int) * N * M);
	hipMalloc((void**)&dev_c,sizeof(int) * N * M);
	hipMemcpy(dev_a,a,sizeof(int)*N*M,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,sizeof(int)*N*M,hipMemcpyHostToDevice);
	dim3 gird((N+15/16),(M+15)/16);
	dim3 block(16,16);
	add<<<gird,block>>>(dev_a,dev_b,dev_c);
	hipMemcpy(c,dev_c,sizeof(int)*N*M,hipMemcpyDeviceToHost);
	print(c,'c');
}

