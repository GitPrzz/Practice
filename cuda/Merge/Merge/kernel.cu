#include "hip/hip_runtime.h"
/*
cuda�鲢����
���ںϲ���ʱ��Ҫ���С����� ���Բ�û���ҵ��ȽϺ��ʵĲ��з�������
1.����������Ū��˺���,ÿ������Ӧһ���߳�,Ȼ����ö��size���й鲢
2.ÿ���̴߳���һ������,Ȼ��ö��size,ͨ��blockIdx����֪����ǰ��������Ϣ,Ȼ���кϲ�����
//������cpu��Ч�ʡ�����
*/

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <Windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
int N;
const int M = 256;
using namespace std;
void print(int *a)
{
	for(int i = 0;i < N;i++)
		cout <<a[i] <<" ";
	cout <<endl;
}
__device__ int Min(int a,int b)
{
	return a < b ? a : b;
}

//ÿ������Ӧһ���߳�
__global__ void cal(int *a,int *t,int len)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	if(id >= len) return ;
	int size = 2;
	while(size  < len * 2)
	{
		if(id % size == 0)
		{
			int st1 = id,ed1 = st1 + size/2; ed1 = Min(ed1,len);
			int st2 = ed1,ed2 = st2 + size/2; ed2 = Min(ed2,len);
			int pos = st1;
			while(st1 < ed1 && st2 < ed2)
			{
				if(a[st1] > a[st2])t[pos++] = a[st2++];
				else  t[pos++] = a[st1++];
			}
			while(st1 < ed1) t[pos ++] = a[st1++];
			while(st2 < ed2) t[pos++] = a[st2 ++];
		}
		__syncthreads();
		a[id] = t[id];
		size <<= 1;
	}
}
//ÿ�������Ӧһ���߳�
__global__ void cal_1(int *a,int *t,int Size,int len)
{
	int id = blockIdx.x;
	if(id >= len ) return ;
	int st1 = id*Size,ed1 = st1 + Size/2; ed1 = Min(ed1,len);
	int st2 = ed1,ed2 = st2 + Size/2; ed2 = Min(ed2,len);
	int pos = st1;
	while(st1 < ed1 && st2 < ed2)
	{
		if(a[st1] > a[st2])t[pos++] = a[st2++];
		else  t[pos++] = a[st1++];
	}
	while(st1 < ed1) t[pos ++] = a[st1++];
	while(st2 < ed2) t[pos++] = a[st2 ++];
	for(int i = st1;i < ed2 ;i ++)
		a[i] = t[i];
}


void debug(int *dev_t)
{
	int *b = (int *)malloc(sizeof(int) * N);
	hipMemcpy(b,dev_t,sizeof(int)*N,hipMemcpyDeviceToHost);
	print(b);
}
void Merge1(int *dev_a,int *dev_b)
{
	int size = 2;
	while(size < 2 * N)
	{
		cal_1<<<(N+size-1)/size,1>>>(dev_a,dev_b,size,N);
		//debug(dev_b);
		size <<= 1;
	}
}



//CPU����
void mergearray(int a[], int first, int mid, int last, int temp[])  
{  
	mid = min(mid,N-1),last = min(last,N-1);
	int i = first, j = mid + 1 ;
	int m = mid,   n = last, k = 0;  
	while (i <= m && j <= n)  
	{  
		if (a[i] <= a[j])  temp[k++] = a[i++];  
		else  temp[k++] = a[j++];  
	}  
	while (i <= m)  temp[k++] = a[i++];  
	while (j <= n)  temp[k++] = a[j++];  
	for (i = 0; i < k; i++)   a[first + i] = temp[i];  
}  
void mergesort(int a[],int temp[])  
{  
	int size = 2;
	while(size < 2 * N)
	{
		for(int i = 0; i * size < N ;i++) mergearray(a,i*size,i*size+size/2-1,i*size + size-1,temp);
		size <<= 1;
	}
}  

void merge(int *a)
{
	int *p = (int *)malloc(sizeof(int) * N);
	mergesort(a, p);
}

void Merge(int *dev_a,int *dev_b)
{
	cout << N <<endl;
	cal<<<(N + M -1)/M,M>>>(dev_a,dev_b,N);
}

void cuda_time_cal1(int *dev_a,int *dev_b)
{	
	float time;
	hipEvent_t s,t;
	hipEventCreate(&s);
	hipEventCreate(&t);
	hipEventRecord(s,0);
	Merge(dev_a,dev_b);
	hipEventRecord(t,0);
	hipEventSynchronize(t);
	hipEventElapsedTime(&time,s,t);
	printf("GPU1 Cost of time:%.5f ms\n",time);
	hipEventDestroy(s);
	hipEventDestroy(t);
}

void cuda_time_cal2(int *dev_a,int *dev_b)
{	
	float time;
	hipEvent_t s,t;
	hipEventCreate(&s);
	hipEventCreate(&t);
	hipEventRecord(s,0);
	Merge1(dev_a,dev_b);
	hipEventRecord(t,0);
	hipEventSynchronize(t);
	hipEventElapsedTime(&time,s,t);
	printf("GPU2 Cost of time:%.5f ms\n",time);
	hipEventDestroy(s);
	hipEventDestroy(t);
}

void cpu_time_cal(int *a)
{	
	LARGE_INTEGER t1,t2,tc;
	QueryPerformanceFrequency(&tc);
	QueryPerformanceCounter(&t1);
	merge(a);
	QueryPerformanceCounter(&t2);
	printf("Use Time:%f\n",(t2.QuadPart - t1.QuadPart)*1000.0/tc.QuadPart);
}

int main()
{
	int *a,*dev_a,*dev_b;
	int *b;
	cout << "�������С��"<<endl;
	cin >> N;
	a = (int *) malloc(sizeof(int) * N);
	b = (int *) malloc(sizeof(int) * N);
	hipMalloc((void**)&dev_a,sizeof(int) *N );
	hipMalloc((void**)&dev_b,sizeof(int) *N );
	srand(time(NULL));
	for(int i = 0;i < N ;i++)
		a[i] = rand()%N;
	//print(a);

	hipMemcpy(dev_a,a,sizeof(int)*N,hipMemcpyHostToDevice);
	cuda_time_cal1(dev_a,dev_b);
	hipMemcpy(b,dev_a,sizeof(int)*N,hipMemcpyDeviceToHost);
	//print(b);

	hipMemcpy(dev_a,a,sizeof(int)*N,hipMemcpyHostToDevice);
	cuda_time_cal2(dev_a,dev_b);
	hipMemcpy(b,dev_a,sizeof(int)*N,hipMemcpyDeviceToHost);
	//print(b);

	return 0;
}
