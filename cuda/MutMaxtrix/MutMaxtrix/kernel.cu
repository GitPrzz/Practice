#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <cstdlib>
#include <stdio.h>
#include <string>
#include <iostream>
using namespace std;
const int block_size = 16;
__global__ void cal(int *a, size_t la, int *b,size_t lb,int *c,size_t lc,int n)
{
	__shared__ int matA[block_size][block_size];
	__shared__ int matB[block_size][block_size];
	const int tidc = threadIdx.x;  
	const int tidr = threadIdx.y;  
	const int bidc = blockIdx.x * block_size;  
	const int bidr = blockIdx.y * block_size;  
	int result = 0;
	for(int i = 0;i < n;i += block_size )
	{
		matA[tidr][tidc] = a[(tidr + bidr) * la +  tidc  + i];
		matB[tidr][tidc] = b[(tidr + i) * lb + tidc + bidc];
		syncthreads();
		for(int j = 0;j < block_size;j++)
		{
			result += matA[tidr][j] * matB[j][tidc];
		}
		syncthreads();	
	}
	c[(tidr + bidr) * lc +  tidc + bidc]  = result;
}

void print(int *a,int ldx,int ldy)
{
	for(int i = 0;i < ldx;i ++)
	{
		for(int j = 0;j < ldy;j++)
		{
			printf("%d ",a[i * ldy + j]);
		}
		printf("\n");
	}
}

void Get_Rand(int *a ,int n,int *b)
{
	srand(NULL);
	for(int i = 0;i < n * n;i++)
	{
		a[i] = rand()% n;
		b[i] = rand()% n;
	}
	/*printf("matA:\n");
	print(a,n,n);
	printf("matB:\n");
	print(b,n,n);*/
}

__global__ void cal1(int *a ,int la,int *b,int lb,int *c,int lc,int n)
{
	extern __shared__ int share[];
	int id = blockIdx.x;
	int tid = threadIdx.x;
	//for(int i = 0;i < la;i++)
	if(tid >= la) return ;
	share[tid] = a[id * la + tid];
	__syncthreads();

	for(int i = 0;i < la;i++)
	{
		c[id*lc + tid] += share[i] * b[i * lb + tid];
	}
}

void Get_ans(int *a,int *b,int la,int lb,int lc)
{
	int *c;
	c = (int*)malloc(sizeof(int) * la * lc);
	memset(c,0,sizeof(int)*la*lc);
	for(int i = 0;i < la;i++)
	{
		for(int j = 0;j < lc;j++)
		{
			for(int k = 0;k < lb;k++)
			{
				c[i*lc+j] += a[i*lb+k] * b[k*lc+j];
			}
		}
	}
	printf("dev_C\n");
	print(c,la,lc);
}

int lda = 4,ldb = 2;
int main()
{
	freopen("out.txt","w",stdout);
	int n = 100;
	int *a,*b,*c;
	int *dev_a,*dev_b,*dev_c;
	a = (int*)malloc(sizeof(int) * n * n);
	b = (int*)malloc(sizeof(int) * n * n);
	c = (int*)malloc(sizeof(int) * n * n);
	Get_Rand(a,n,b);
	Get_ans(a,b,n,n,n);
	
	//print(a,n,n);
	size_t pitch_a,pitch_b,pitch_c;
	hipMallocPitch((void**)&dev_a,&pitch_a,sizeof(int)*n,n);
	hipMallocPitch((void**)&dev_b,&pitch_b,sizeof(int)*n,n);
	hipMallocPitch((void**)&dev_c,&pitch_c,sizeof(int)*n,n);
	hipMemcpy2D(dev_a,pitch_a,a,sizeof(int)*n,sizeof(int) * n,n,hipMemcpyHostToDevice);
	hipMemcpy2D(dev_b,pitch_b,b,sizeof(int)*n,sizeof(int) * n,n,hipMemcpyHostToDevice);
	int tx = (n + block_size -1) / block_size;
	cout << "pitch"<< pitch_a <<endl;
	cal1<<<n,block_size,block_size>>>(dev_a, pitch_a /sizeof(int),dev_b,pitch_b / sizeof(int),dev_c,pitch_c/sizeof(int),n);
	hipMemcpy2D(c,sizeof(int)*n,dev_c,pitch_c,sizeof(int)*n,n,hipMemcpyDeviceToHost);
	printf("matC\n");
	print(c,n,n);
	return 0;
}

